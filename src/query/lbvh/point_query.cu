#include "hip/hip_runtime.h"
#include "point_query.h"

#include "lbvh.cuh"
#include "rtspatial/utils/queue.h"
#include "stopwatch.h"
struct aabb_getter {
  __device__ lbvh::aabb<float> operator()(float4 &box) const noexcept {
    lbvh::aabb<float> retval;

    retval.lower = make_float4(box.x, box.y, 0, 0);
    retval.upper = make_float4(box.z, box.w, 0, 0);
    return retval;
  }
};

time_stat RunPointQueryLBVH(const std::vector<box_t> &boxes,
                            const std::vector<point_t> &queries,
                            const BenchmarkConfig &config) {
  std::vector<float4> corners;

  corners.reserve(boxes.size());
  for (auto &box : boxes) {
    corners.push_back(make_float4(box.min_corner().x(), box.min_corner().y(),
                                  box.max_corner().x(), box.max_corner().y()));
  }

  thrust::device_vector<float4> d_boxes(corners);

  corners.clear();
  for (auto &p : queries) {
    corners.push_back(make_float4(p.x(), p.y(), p.x(), p.y()));
  }
  thrust::device_vector<float4> d_queries(corners);
  lbvh::bvh<coord_t, float4, aabb_getter> lbvh;
  time_stat ts;
  Stopwatch sw;

  ts.num_geoms = boxes.size();
  ts.num_queries = queries.size();

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    sw.start();
    lbvh.assign(d_boxes.begin(), d_boxes.end());
    sw.stop();
    ts.insert_ms.push_back(sw.ms());
  }
  d_boxes.resize(0);
  d_boxes.shrink_to_fit();

  auto p_lbvh = lbvh.get_device_repr();

#ifdef COLLECT_RESULTS
  rtspatial::Queue<thrust::pair<uint32_t, uint32_t>> results;
  results.Init(std::max(
      1ul, (size_t)(boxes.size() * queries.size() * config.load_factor)));
  auto d_results = results.DeviceObject();
#endif

#ifdef COUNT_RESULTS
  rtspatial::SharedValue<unsigned long long int> counter;
  auto *p_counter = counter.data();
#endif

  for (int i = 0; i < config.warmup + config.repeat; i++) {
#ifdef COLLECT_RESULTS
    results.Clear();
#endif

#ifdef COUNT_RESULTS
    counter.set(0);
#endif

    sw.start();
    switch (config.query_type) {
    case BenchmarkConfig::QueryType::kPointContains: {
      thrust::for_each(
          thrust::make_zip_iterator(thrust::make_tuple(
              thrust::make_counting_iterator<uint32_t>(0), d_queries.begin())),
          thrust::make_zip_iterator(thrust::make_tuple(
              thrust::make_counting_iterator<uint32_t>(d_queries.size()),
              d_queries.end())),
          [=] __device__(const thrust::tuple<uint32_t, float4> &tuple) mutable {
            uint32_t query_id = thrust::get<0>(tuple);
            const auto &query = thrust::get<1>(tuple);

            lbvh::aabb<float> box;

            box.lower = make_float4(query.x, query.y, 0, 0);
            box.upper = make_float4(query.z, query.w, 0, 0);

            lbvh::query_device_all(
                p_lbvh, lbvh::overlaps(box),
                [=] __device__(std::uint32_t geom_id) mutable {
#ifdef COLLECT_RESULTS
                  d_results.Append(thrust::make_pair(geom_id, query_id));
#endif

#ifdef COUNT_RESULTS
                  atomicAdd(p_counter, 1);
#endif
                });
          });

      break;
    }
    default:
      abort();
    }
    // Implicit barrier
#ifdef COLLECT_RESULTS
    ts.num_results = results.size();
#endif

#ifdef COUNT_RESULTS
    ts.num_results = counter.get();
#endif
    sw.stop();
    ts.query_ms.push_back(sw.ms());
  }

  return ts;
}