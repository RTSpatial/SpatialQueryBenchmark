#include "hip/hip_runtime.h"
#include "lbvh.cuh"
#include "range_query.h"
#include "rtspatial/utils/queue.h"
#include "stopwatch.h"

struct aabb_getter {
  __device__ lbvh::aabb<float> operator()(float4 &box) const noexcept {
    lbvh::aabb<float> retval;

    retval.lower = make_float4(box.x, box.y, 0, 0);
    retval.upper = make_float4(box.z, box.w, 0, 0);
    return retval;
  }
};

time_stat RunRangeQueryLBVH(const std::vector<box_t> &boxes,
                            const std::vector<box_t> &queries,
                            const BenchmarkConfig &config) {
  std::vector<float4> corners;

  corners.reserve(boxes.size());
  for (auto &box : boxes) {
    corners.push_back(make_float4(box.min_corner().x(), box.min_corner().y(),
                                  box.max_corner().x(), box.max_corner().y()));
  }

  thrust::device_vector<float4> d_boxes(corners);

  corners.clear();
  for (auto &box : queries) {
    corners.push_back(make_float4(box.min_corner().x(), box.min_corner().y(),
                                  box.max_corner().x(), box.max_corner().y()));
  }

  thrust::device_vector<float4> d_queries(corners);
  lbvh::bvh<coord_t, float4, aabb_getter> lbvh;
  time_stat ts;
  Stopwatch sw;

  ts.num_geoms = boxes.size();
  ts.num_queries = queries.size();

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    sw.start();
    lbvh.assign(d_boxes.begin(), d_boxes.end());
    sw.stop();
    ts.insert_ms.push_back(sw.ms());
  }

  auto p_lbvh = lbvh.get_device_repr();

  rtspatial::Queue<thrust::pair<uint32_t, uint32_t>> results;
  results.Init(std::max(
      1ul, (size_t)(boxes.size() * queries.size() * config.load_factor)));
  auto d_results = results.DeviceObject();

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    results.Clear();
    sw.start();
    switch (config.query_type) {
    case BenchmarkConfig::QueryType::kRangeContains: {
      auto *p_boxes = thrust::raw_pointer_cast(d_boxes.data());

      thrust::for_each(
          thrust::make_zip_iterator(thrust::make_tuple(
              thrust::make_counting_iterator<uint32_t>(0), d_queries.begin())),
          thrust::make_zip_iterator(thrust::make_tuple(
              thrust::make_counting_iterator<uint32_t>(d_queries.size()),
              d_queries.end())),
          [=] __device__(const thrust::tuple<uint32_t, float4> &tuple) mutable {
            uint32_t query_id = thrust::get<0>(tuple);
            const auto &query = thrust::get<1>(tuple);

            lbvh::aabb<float> box;

            box.lower = make_float4(query.x, query.y, 0, 0);
            box.upper = make_float4(query.z, query.w, 0, 0);

            lbvh::query_device_all(
                p_lbvh, lbvh::contains(box),
                [=] __device__(std::uint32_t geom_id) mutable {
                  d_results.Append(thrust::make_pair(geom_id, query_id));
                });
          });
      break;
    }
    case BenchmarkConfig::QueryType::kRangeIntersects: {
      thrust::for_each(
          thrust::make_zip_iterator(thrust::make_tuple(
              thrust::make_counting_iterator<uint32_t>(0), d_queries.begin())),
          thrust::make_zip_iterator(thrust::make_tuple(
              thrust::make_counting_iterator<uint32_t>(d_queries.size()),
              d_queries.end())),
          [=] __device__(const thrust::tuple<uint32_t, float4> &tuple) mutable {
            uint32_t query_id = thrust::get<0>(tuple);
            const auto &query = thrust::get<1>(tuple);

            lbvh::aabb<float> box;

            box.lower = make_float4(query.x, query.y, 0, 0);
            box.upper = make_float4(query.z, query.w, 0, 0);

            lbvh::query_device_all(
                p_lbvh, lbvh::overlaps(box),
                [=] __device__(std::uint32_t geom_id) mutable {
                  d_results.Append(thrust::make_pair(geom_id, query_id));
                });
          });
      break;
    }
    default:
      abort();
    }
    // Implicit barrier
    ts.num_results = results.size();
    sw.stop();
    ts.query_ms.push_back(sw.ms());
  }

  return ts;
}
