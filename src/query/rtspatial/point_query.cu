#include "hip/hip_runtime.h"
#include "point_query.h"
#include "query/rtspatial/common.h"
#include "rtspatial/rtspatial.h"
#include "stopwatch.h"
time_stat RunPointQueryRTSpatial(const std::vector<box_t> &boxes,
                                 const std::vector<point_t> &queries,
                                 const BenchmarkConfig &config) {
  rtspatial::Stream stream;
  rtspatial::SpatialIndex<coord_t, 2, true> index;
  thrust::device_vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>
      d_boxes;
  thrust::device_vector<rtspatial::Point<coord_t, 2>> d_queries;
  rtspatial::Config idx_config;

  idx_config.ptx_root = std::string(RTSPATIAL_PTX_DIR);

  CopyBoxes(boxes, d_boxes);
  CopyPoints(queries, d_queries);

  index.Init(idx_config);
  time_stat ts;
  Stopwatch sw;

  ts.num_geoms = boxes.size();
  ts.num_queries = queries.size();

  rtspatial::Queue<thrust::pair<uint32_t, uint32_t>> results;
  rtspatial::SharedValue<
      rtspatial::Queue<thrust::pair<uint32_t, uint32_t>>::device_t>
      d_results;

  results.Init(std::max(
      1ul, (size_t)(boxes.size() * queries.size() * config.load_factor)));
  d_results.set(stream.hip_stream(), results.DeviceObject());

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    index.Clear();
    sw.start();
    index.Insert(d_boxes, stream.hip_stream());
    stream.Sync();
    sw.stop();
    ts.insert_ms.push_back(sw.ms());
  }
  d_boxes.resize(0);
  d_boxes.shrink_to_fit();

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    results.Clear(stream.hip_stream());
    sw.start();
    switch (config.query_type) {
    case BenchmarkConfig::QueryType::kPointContains: {
      index.ContainsWhatQuery(d_queries, d_results.data(),
                              stream.hip_stream());
      break;
    }
    default:
      abort();
    }
    // Implicit barrier
    ts.num_results = results.size(stream.hip_stream());
    sw.stop();
    ts.query_ms.push_back(sw.ms());
  }

  return ts;
}