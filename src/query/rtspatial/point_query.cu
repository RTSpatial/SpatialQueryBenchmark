#include "hip/hip_runtime.h"
#include "point_query.h"
#include "query/rtspatial/common.h"
#include "rtspatial/rtspatial.h"
#include "stopwatch.h"
time_stat RunPointQueryRTSpatial(const std::vector<box_t> &boxes,
                                 const std::vector<point_t> &queries,
                                 const BenchmarkConfig &config) {
  rtspatial::Stream stream;
  rtspatial::SpatialIndex<coord_t, 2> index;
  thrust::device_vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>
      d_boxes;
  thrust::device_vector<rtspatial::Point<coord_t, 2>> d_queries;
  rtspatial::Config idx_config;

  idx_config.ptx_root = std::string(RTSPATIAL_PTX_DIR);
  idx_config.max_geometries = boxes.size();

  CopyBoxes(boxes, d_boxes);
  CopyPoints(queries, d_queries);

  index.Init(idx_config);
  time_stat ts;
  Stopwatch sw;

  ts.num_geoms = boxes.size();
  ts.num_queries = queries.size();

  rtspatial::Queue<thrust::pair<uint32_t, uint32_t>> results;
  rtspatial::SharedValue<
      rtspatial::Queue<thrust::pair<uint32_t, uint32_t>>::device_t>
      d_results;

  results.Init(std::max(
      1ul, (size_t)(boxes.size() * queries.size() * config.load_factor)));
  d_results.set(stream.hip_stream(), results.DeviceObject());

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    index.Clear();
    sw.start();
    index.Insert(
        rtspatial::ArrayView<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>(
            d_boxes),
        stream.hip_stream());
    stream.Sync();
    sw.stop();
    ts.insert_ms.push_back(sw.ms());
  }

  auto updates = GenerateUpdates(boxes, config.update_ratio);

  auto run_queries = [&](std::vector<double> &running_times) {
    for (int i = 0; i < config.warmup + config.repeat; i++) {
      results.Clear(stream.hip_stream());
      sw.start();
      switch (config.query_type) {
      case BenchmarkConfig::QueryType::kPointContains: {
        index.Query(rtspatial::Predicate::kContains, d_queries,
                    d_results.data(), stream.hip_stream());
        break;
      }
      default:
        abort();
      }
      // Implicit barrier
      ts.num_results = results.size(stream.hip_stream());
      sw.stop();
      running_times.push_back(sw.ms());
    }
  };

  if (!updates.empty()) {
    index.Update(
        rtspatial::ArrayView<thrust::pair<
            size_t, rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>>(
            updates),
        stream.hip_stream());
    stream.Sync();

    // Run Query after updates
    run_queries(ts.query_ms_after_update);

    UpdateBoxes(d_boxes, updates);
    // Rebuild Index on updated geometries
    index.Clear();
    index.Insert(
        rtspatial::ArrayView<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>(
            d_boxes),
        stream.hip_stream());
  }

  run_queries(ts.query_ms);
  return ts;
}