#include "hip/hip_runtime.h"
#include "point_query.h"
#include "query/rtspatial/common.h"
#include "rtspatial/rtspatial.h"
#include "stopwatch.h"
time_stat RunPointQueryRTSpatial(const std::vector<box_t> &boxes,
                                 const std::vector<point_t> &queries,
                                 const BenchmarkConfig &config) {
  rtspatial::Stream stream;
  rtspatial::SpatialIndex<coord_t, 2, true> index;
  rtspatial::Queue<thrust::pair<uint32_t, uint32_t>> results;
  thrust::device_vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>
      d_boxes;
  thrust::device_vector<rtspatial::Point<coord_t, 2>> d_queries;
  rtspatial::Config idx_config;

  idx_config.ptx_root = std::string(RTSPATIAL_LIBRARY_DIR) + "/ptx";

  CopyBoxes(boxes, d_boxes);
  CopyPoints(queries, d_queries);

  index.Init(idx_config);
  results.Init(std::max(
      1ul, (size_t)(boxes.size() * queries.size() * config.load_factor)));
  results.Clear(stream.hip_stream());
  time_stat ts;
  Stopwatch sw;

  ts.num_geoms = boxes.size();
  ts.num_queries = queries.size();

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    index.Clear();
    sw.start();
    index.Insert(d_boxes, stream.hip_stream());
    stream.Sync();
    sw.stop();
    ts.insert_ms.push_back(sw.ms());
  }

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    results.Clear(stream.hip_stream());
    sw.start();
    switch (config.query_type) {
    case BenchmarkConfig::QueryType::kPointContains: {
      index.ContainsWhatQuery(d_queries, results, stream.hip_stream());
      break;
    }
    default:
      abort();
    }
    ts.num_results = results.size(stream.hip_stream());
    sw.stop();
    ts.query_ms.push_back(sw.ms());
  }

  return ts;
}