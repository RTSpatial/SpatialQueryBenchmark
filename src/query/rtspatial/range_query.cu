#include "hip/hip_runtime.h"
#include "query/rtspatial/common.h"
#include "range_query.h"
#include "rtspatial/rtspatial.h"
#include "stopwatch.h"

time_stat RunRangeQueryRTSpatial(const std::vector<box_t> &boxes,
                                 const std::vector<box_t> &queries,
                                 const BenchmarkConfig &config) {
  rtspatial::Stream stream;
  rtspatial::SpatialIndex<coord_t, 2, false> index;
  thrust::device_vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>
      d_boxes, d_queries;
  rtspatial::Config idx_config;

  idx_config.ptx_root = std::string(RTSPATIAL_PTX_DIR);

  CopyBoxes(boxes, d_boxes);
  CopyBoxes(queries, d_queries);

  index.Init(idx_config);
  time_stat ts;
  Stopwatch sw;

  ts.num_geoms = boxes.size();
  ts.num_queries = queries.size();

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    index.Clear();
    sw.start();
    index.Insert(d_boxes, stream.hip_stream());
    stream.Sync();
    sw.stop();
    ts.insert_ms.push_back(sw.ms());
  }
  d_boxes.resize(0);
  d_boxes.shrink_to_fit();

  rtspatial::Queue<thrust::pair<uint32_t, uint32_t>> results;
  rtspatial::SharedValue<
      rtspatial::Queue<thrust::pair<uint32_t, uint32_t>>::device_t>
      d_results;

  results.Init(std::max(
      1ul, (size_t)(boxes.size() * queries.size() * config.load_factor)));
  d_results.set(stream.hip_stream(), results.DeviceObject());

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    results.Clear(stream.hip_stream());
    sw.start();
    switch (config.query_type) {
    case BenchmarkConfig::QueryType::kRangeContains: {
      index.ContainsWhatQuery(d_queries, d_results.data(),
                              stream.hip_stream());
      break;
    }
    case BenchmarkConfig::QueryType::kRangeIntersects: {
      int best_parallelism =
          index.CalculateBestParallelism(d_queries, stream.hip_stream());
      index.IntersectsWhatQuery(d_queries, d_results.data(),
                                stream.hip_stream(), best_parallelism);
      break;
    }
    default:
      abort();
    }
    // Implicit barrier
    ts.num_results = results.size(stream.hip_stream());
    sw.stop();
    ts.query_ms.push_back(sw.ms());
  }

  return ts;
}

time_stat
RunRangeQueryRTSpatialVaryParallelism(const std::vector<box_t> &boxes,
                                      const std::vector<box_t> &queries,
                                      const BenchmarkConfig &config) {
  rtspatial::Stream stream;
  rtspatial::SpatialIndex<coord_t, 2, false> index;
  thrust::device_vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>
      d_boxes, d_queries;
  rtspatial::Config idx_config;

  idx_config.ptx_root = std::string(RTSPATIAL_PTX_DIR);

  CopyBoxes(boxes, d_boxes);
  CopyBoxes(queries, d_queries);

  index.Init(idx_config);
  time_stat ts;
  Stopwatch sw;

  ts.num_geoms = boxes.size();
  ts.num_queries = queries.size();

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    index.Clear();
    sw.start();
    index.Insert(d_boxes, stream.hip_stream());
    stream.Sync();
    sw.stop();
    ts.insert_ms.push_back(sw.ms());
  }
  d_boxes.resize(0);
  d_boxes.shrink_to_fit();

  rtspatial::Queue<thrust::pair<uint32_t, uint32_t>> results;
  rtspatial::SharedValue<
      rtspatial::Queue<thrust::pair<uint32_t, uint32_t>>::device_t>
      d_results;

  results.Init(std::max(
      1ul, (size_t)(boxes.size() * queries.size() * config.load_factor)));
  d_results.set(stream.hip_stream(), results.DeviceObject());

  for (int i = 1; i <= config.parallelism; i *= 2) {
    results.Clear(stream.hip_stream());
    sw.start();
    index.IntersectsWhatQuery(d_queries, d_results.data(), stream.hip_stream(),
                              i);
    ts.num_results = results.size(stream.hip_stream());
    sw.stop();
    ts.query_ms.push_back(sw.ms());
  }

  return ts;
}