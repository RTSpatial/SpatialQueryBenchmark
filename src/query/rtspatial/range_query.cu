#include "hip/hip_runtime.h"
#include "query/rtspatial/common.h"
#include "range_query.h"
#include "rtspatial/rtspatial.h"
#include "stopwatch.h"

time_stat RunRangeQueryRTSpatial(const std::vector<box_t> &boxes,
                                 const std::vector<box_t> &queries,
                                 const BenchmarkConfig &config) {
  rtspatial::Stream stream;
  rtspatial::SpatialIndex<coord_t, 2> index;
  thrust::device_vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>
      d_boxes, d_queries;
  rtspatial::Config idx_config;

  idx_config.ptx_root = std::string(RTSPATIAL_PTX_DIR);
  idx_config.intersect_cost_weight = 0.90;
  idx_config.max_geometries = boxes.size();
  idx_config.compact = false;

  CopyBoxes(boxes, d_boxes);
  CopyBoxes(queries, d_queries);

  index.Init(idx_config);
  time_stat ts;
  Stopwatch sw;

  ts.num_geoms = boxes.size();
  ts.num_queries = queries.size();
  auto queue_size = std::max(
      1ul, (size_t)(boxes.size() * queries.size() * config.load_factor));

  std::cout << "Queue size "
            << queue_size * sizeof(thrust::pair<uint32_t, uint32_t>) / 1024 /
                   1024
            << " MB" << std::endl;

  rtspatial::Queue<thrust::pair<uint32_t, uint32_t>> results;
  rtspatial::SharedValue<
      rtspatial::Queue<thrust::pair<uint32_t, uint32_t>>::device_t>
      d_results;

  results.Init(queue_size);
  d_results.set(stream.hip_stream(), results.DeviceObject());

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    index.Clear();
    sw.start();
    index.Insert(
        rtspatial::ArrayView<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>(
            d_boxes),
        stream.hip_stream());
    stream.Sync();
    sw.stop();
    ts.insert_ms.push_back(sw.ms());
  }

  index.PrintMemoryUsage();

  auto updates = GenerateUpdates(boxes, config.update_ratio);

  auto run_queries = [&](std::vector<double> &running_times) {
    for (int i = 0; i < config.warmup + config.repeat; i++) {
      results.Clear(stream.hip_stream());
      sw.start();
      switch (config.query_type) {
      case BenchmarkConfig::QueryType::kRangeContains: {
        index.Query(rtspatial::Predicate::kContains, d_queries,
                    d_results.data(), stream.hip_stream());
        break;
      }
      case BenchmarkConfig::QueryType::kRangeIntersects: {
        index.Query(rtspatial::Predicate::kIntersects, d_queries,
                    d_results.data(), stream.hip_stream());
        break;
      }
      default:
        abort();
      }
      // Implicit barrier
      ts.num_results = results.size(stream.hip_stream());
      sw.stop();
      running_times.push_back(sw.ms());
    }
  };

  if (!updates.empty()) {
    index.Update(
        rtspatial::ArrayView<thrust::pair<
            size_t, rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>>(
            updates),
        stream.hip_stream());
    stream.Sync();

    // Run Query after updates
    run_queries(ts.query_ms_after_update);

    UpdateBoxes(d_boxes, updates);
    // Rebuild Index on updated geometries
    index.Clear();
    index.Insert(
        rtspatial::ArrayView<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>(
            d_boxes),
        stream.hip_stream());
  }

  run_queries(ts.query_ms);

  return ts;
}

time_stat
RunRangeQueryRTSpatialVaryParallelism(const std::vector<box_t> &boxes,
                                      const std::vector<box_t> &queries,
                                      const BenchmarkConfig &config) {
  rtspatial::Stream stream;
  rtspatial::SpatialIndex<coord_t, 2> index;
  thrust::device_vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>
      d_boxes, d_queries;
  rtspatial::Config idx_config;

  idx_config.ptx_root = std::string(RTSPATIAL_PTX_DIR);
  idx_config.intersect_cost_weight = 0.90;
  idx_config.prefer_fast_build_query = false;

  CopyBoxes(boxes, d_boxes);
  CopyBoxes(queries, d_queries);

  index.Init(idx_config);
  time_stat ts;
  Stopwatch sw;

  ts.num_geoms = boxes.size();
  ts.num_queries = queries.size();

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    index.Clear();
    sw.start();
    index.Insert(
        rtspatial::ArrayView<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>(
            d_boxes),
        stream.hip_stream());
    stream.Sync();
    sw.stop();
    ts.insert_ms.push_back(sw.ms());
  }

  d_boxes.resize(0);
  d_boxes.shrink_to_fit();

  rtspatial::Queue<thrust::pair<uint32_t, uint32_t>> results;
  rtspatial::SharedValue<
      rtspatial::Queue<thrust::pair<uint32_t, uint32_t>>::device_t>
      d_results;

  results.Init(std::max(
      1ul, (size_t)(boxes.size() * queries.size() * config.load_factor)));
  d_results.set(stream.hip_stream(), results.DeviceObject());

  for (int i = 1; i <= config.parallelism; i *= 2) {
    results.Clear(stream.hip_stream());
    sw.start();
    index.IntersectsWhatQueryProfiling(d_queries, d_results.data(),
                                       stream.hip_stream(), i);
    ts.num_results = results.size(stream.hip_stream());
    sw.stop();
    ts.query_ms.push_back(sw.ms());
  }

  sw.start();
  int pred = index.CalculateBestParallelism(d_queries, stream.hip_stream());
  sw.stop();
  std::cout << "Predicated Parallelism " << pred << " Time " << sw.ms() << " ms"
            << std::endl;

  results.Clear(stream.hip_stream());

  index.IntersectsWhatQueryProfiling(d_queries, d_results.data(),
                                     stream.hip_stream(), pred);

  return ts;
}