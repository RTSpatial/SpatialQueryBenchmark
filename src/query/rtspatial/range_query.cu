#include "hip/hip_runtime.h"
#include "query/rtspatial/common.h"
#include "range_query.h"
#include "rtspatial/rtspatial.h"
#include "stopwatch.h"

time_stat RunRangeQueryRTSpatial(const std::vector<box_t> &boxes,
                                 const std::vector<box_t> &queries,
                                 const BenchmarkConfig &config) {
  rtspatial::Stream stream;
  rtspatial::SpatialIndex<coord_t, 2, false> index;
  rtspatial::Queue<thrust::pair<uint32_t, uint32_t>> results;
  thrust::device_vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>
      d_boxes, d_queries;
  rtspatial::Config idx_config;

  idx_config.ptx_root = std::string(RTSPATIAL_LIBRARY_DIR) + "/ptx";

  CopyBoxes(boxes, d_boxes);
  CopyBoxes(queries, d_queries);

  index.Init(idx_config);
  time_stat ts;
  Stopwatch sw;

  ts.num_geoms = boxes.size();
  ts.num_queries = queries.size();

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    index.Clear();
    sw.start();
    index.Insert(d_boxes, stream.hip_stream());
    stream.Sync();
    sw.stop();
    ts.insert_ms.push_back(sw.ms());
  }
  d_boxes.resize(0);
  d_boxes.shrink_to_fit();
  size_t queue_size = std::max(
      1ul, (size_t)(boxes.size() * queries.size() * config.load_factor));

  std::cout << "Result queue capacity: " << queue_size << ", memory: "
            << queue_size * sizeof(thrust::pair<uint32_t, uint32_t>) / 1024 /
                   1024
            << " MB" << std::endl;
  results.Init(queue_size);
  results.Clear(stream.hip_stream());

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    results.Clear(stream.hip_stream());
    sw.start();
    switch (config.query_type) {
    case BenchmarkConfig::QueryType::kRangeContains: {
      index.ContainsWhatQuery(d_queries, results, stream.hip_stream());
      break;
    }
    case BenchmarkConfig::QueryType::kRangeIntersects: {
      index.IntersectsWhatQuery(d_queries, results, stream.hip_stream(),
                                FLAGS_rays);
      break;
    }
    default:
      abort();
    }
    ts.num_results = results.size(stream.hip_stream());
    sw.stop();
    ts.query_ms.push_back(sw.ms());
  }

  return ts;
}