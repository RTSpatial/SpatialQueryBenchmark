#include "hip/hip_runtime.h"
#include "query/rtspatial/common.h"
#include "range_query.h"
#include "rtspatial/rtspatial.h"
#include "stopwatch.h"

time_stat RunRangeQueryRTSpatial(const std::vector<box_t> &boxes,
                                 const std::vector<box_t> &queries,
                                 const BenchmarkConfig &config) {
  rtspatial::Stream stream;
  rtspatial::SpatialIndex<coord_t, 2, false> index;
  thrust::device_vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>
      d_boxes, d_queries;
  rtspatial::Config idx_config;

  idx_config.ptx_root = std::string(RTSPATIAL_PTX_DIR);

  CopyBoxes(boxes, d_boxes);
  CopyBoxes(queries, d_queries);

  index.Init(idx_config);
  time_stat ts;
  Stopwatch sw;

  ts.num_geoms = boxes.size();
  ts.num_queries = queries.size();

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    index.Clear();
    sw.start();
    index.Insert(d_boxes, stream.hip_stream());
    stream.Sync();
    sw.stop();
    ts.insert_ms.push_back(sw.ms());
  }
  d_boxes.resize(0);
  d_boxes.shrink_to_fit();

#ifdef COLLECT_RESULTS
  rtspatial::Queue<thrust::pair<uint32_t, uint32_t>> results;
  rtspatial::SharedValue<
      rtspatial::Queue<thrust::pair<uint32_t, uint32_t>>::device_t>
      d_results;

  results.Init(std::max(
      1ul, (size_t)(boxes.size() * queries.size() * config.load_factor)));
  d_results.set(stream.hip_stream(), results.DeviceObject());
#endif

#ifdef COUNT_RESULTS
  rtspatial::SharedValue<unsigned long long int> counter;
#endif

  for (int i = 0; i < config.warmup + config.repeat; i++) {
#ifdef COLLECT_RESULTS
    results.Clear(stream.hip_stream());
#endif

#ifdef COUNT_RESULTS
    counter.set(stream.hip_stream(), 0);
#endif

    sw.start();
    switch (config.query_type) {
    case BenchmarkConfig::QueryType::kRangeContains: {
#ifdef COLLECT_RESULTS
      index.ContainsWhatQuery(d_queries, d_results.data(),
                              stream.hip_stream());
#endif

#ifdef COUNT_RESULTS
      index.ContainsWhatQuery(d_queries, counter.data(), stream.hip_stream());
#endif
      break;
    }
    case BenchmarkConfig::QueryType::kRangeIntersects: {
      int best_parallelism =
          index.CalculateBestParallelism(d_queries, stream.hip_stream());
#ifdef COLLECT_RESULTS
      index.IntersectsWhatQuery(d_queries, d_results.data(),
                                stream.hip_stream(), best_parallelism);
#endif

#ifdef COUNT_RESULTS
      index.IntersectsWhatQuery(d_queries, counter.data(), stream.hip_stream(),
                                best_parallelism);
#endif
      break;
    }
    default:
      abort();
    }
    // Implicit barrier
#ifdef COLLECT_RESULTS
    ts.num_results = results.size(stream.hip_stream());
#endif

#ifdef COUNT_RESULTS
    ts.num_results = counter.get(stream.hip_stream());
#endif
    sw.stop();
    ts.query_ms.push_back(sw.ms());
  }

  return ts;
}

time_stat
RunRangeQueryRTSpatialVaryParallelism(const std::vector<box_t> &boxes,
                                      const std::vector<box_t> &queries,
                                      const BenchmarkConfig &config) {
  rtspatial::Stream stream;
  rtspatial::SpatialIndex<coord_t, 2, false> index;
  rtspatial::SharedValue<unsigned long long int> counter;
  thrust::device_vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>
      d_boxes, d_queries;
  rtspatial::Config idx_config;

  idx_config.ptx_root = std::string(RTSPATIAL_PTX_DIR);

  CopyBoxes(boxes, d_boxes);
  CopyBoxes(queries, d_queries);

  index.Init(idx_config);
  time_stat ts;
  Stopwatch sw;

  ts.num_geoms = boxes.size();
  ts.num_queries = queries.size();

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    index.Clear();
    sw.start();
    index.Insert(d_boxes, stream.hip_stream());
    stream.Sync();
    sw.stop();
    ts.insert_ms.push_back(sw.ms());
  }
  d_boxes.resize(0);
  d_boxes.shrink_to_fit();

  for (int i = 1; i <= config.parallelism; i *= 2) {
    counter.set(stream.hip_stream(), 0);
    sw.start();
    index.IntersectsWhatQuery(d_queries, counter.data(), stream.hip_stream(),
                              i);
    ts.num_results = counter.get(stream.hip_stream());
    sw.stop();
    ts.query_ms.push_back(sw.ms());
  }

  return ts;
}