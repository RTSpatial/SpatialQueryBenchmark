#include "hip/hip_runtime.h"
#include "cdb_loader.h"
#include "query/rtspatial/common.h"
#include "query/rtspatial/pip_context.h"
#include "query/rtspatial/pip_query.h"
#include "rtspatial/rtspatial.h"
#include "stopwatch.h"

float next_float_from_double(double v, int dir, int iter = 1) {
  assert(dir == 1 || dir == -1);
  auto fv = static_cast<float>(v); // pos number
  if (fv == 0) {
    return 0.0f;
  }
  float to = v * dir < 0 ? 0 : dir * std::numeric_limits<float>::infinity();

  for (int i = 0; i < iter; i++) {
    fv = std::nextafter(fv, to);
  }

  return fv;
};

time_stat RunPIPQueryRTSpatial(const std::vector<polygon_t> &polygons,
                               const std::vector<point_t> &points,
                               const BenchmarkConfig &config) {
  std::vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>> boxes(
      polygons.size());
  std::vector<rtspatial::Point<coord_t, 2>> queries(points.size());
  std::vector<uint32_t> row_offsets;
  std::vector<float2> vertices;
  uint32_t tail = 0;

  row_offsets.push_back(tail);

  for (size_t i = 0; i < boxes.size(); i++) {
    const auto &polygon = polygons[i];
    coord_t lows[2] = {std::numeric_limits<coord_t>::max(),
                       std::numeric_limits<coord_t>::max()};
    coord_t highs[2] = {std::numeric_limits<coord_t>::lowest(),
                        std::numeric_limits<coord_t>::lowest()};

    for (auto &p : polygon.outer()) {
      lows[0] = std::min(lows[0], p.x());
      highs[0] = std::max(highs[0], p.x());
      lows[1] = std::min(lows[1], p.y());
      highs[1] = std::max(highs[1], p.y());
    }

    rtspatial::Envelope<rtspatial::Point<coord_t, 2>> envelope(
        rtspatial::Point<coord_t, 2>(lows[0], lows[1]),
        rtspatial::Point<coord_t, 2>(highs[0], highs[1]));

    boxes[i] = envelope;

    // https://wrfranklin.org/Research/Short_Notes/pnpoly.html
    vertices.push_back(float2{0, 0});
    tail++;

    for (auto &p : polygon.outer()) {
      vertices.push_back(float2{p.x(), p.y()});
      tail++;
    }
    vertices.push_back(float2{0, 0});
    tail++;

    // fill holes
    for (auto &inner : polygon.inners()) {
      for (auto &p : inner) {
        vertices.push_back(float2{p.x(), p.y()});
        tail++;
      }
      vertices.push_back(float2{0, 0});
      tail++;
    }
    row_offsets.push_back(tail);
  }

  for (size_t i = 0; i < points.size(); i++) {
    queries[i].set_x(points[i].x());
    queries[i].set_y(points[i].y());
  }

  thrust::device_vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>
      d_boxes = boxes;
  thrust::device_vector<rtspatial::Point<coord_t, 2>> d_queries = queries;
  thrust::device_vector<uint32_t> d_row_offsets = row_offsets;
  thrust::device_vector<float2> d_vertices = vertices;

  rtspatial::Stream stream;
  rtspatial::SpatialIndex<float, 2> index;
  rtspatial::Config idx_config;

  idx_config.max_geometries = d_boxes.size();
  idx_config.max_queries = d_queries.size();
  idx_config.ptx_root = std::string(RTSPATIAL_PTX_DIR);

  index.Init(idx_config);
  time_stat ts;
  Stopwatch sw;

  ts.num_geoms = d_boxes.size();
  ts.num_queries = d_queries.size();

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    index.Clear();
    sw.start();
    index.Insert(
        rtspatial::ArrayView<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>(
            d_boxes),
        stream.hip_stream());
    stream.Sync();
    sw.stop();
    ts.insert_ms.push_back(sw.ms());
  }

  rtspatial::Queue<thrust::pair<uint32_t, uint32_t>> results;
  PIPContext ctx;
  rtspatial::SharedValue<PIPContext> d_ctx;

  results.Init(std::max(
      1ul, (size_t)(ts.num_geoms * ts.num_queries * config.load_factor)));
  ctx.row_offsets = d_row_offsets;
  ctx.vertices = d_vertices;
  ctx.points = d_queries;
  ctx.results = results.DeviceObject();

  d_ctx.set(stream.hip_stream(), ctx);

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    results.Clear(stream.hip_stream());
    sw.start();
    index.Query(rtspatial::Predicate::kContains, d_queries, d_ctx.data(),
                stream.hip_stream());
    ts.num_results = results.size(stream.hip_stream());
    sw.stop();
    ts.query_ms.push_back(sw.ms());
  }

  return ts;
}
