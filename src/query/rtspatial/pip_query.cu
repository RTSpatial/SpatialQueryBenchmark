#include "hip/hip_runtime.h"
#include "cdb_loader.h"
#include "query/rtspatial/common.h"
#include "query/rtspatial/lsi_query.h"
#include "rtspatial/rtspatial.h"
#include "stopwatch.h"

float next_float_from_double(double v, int dir, int iter = 1) {
  assert(dir == 1 || dir == -1);
  auto fv = static_cast<float>(v); // pos number
  if (fv == 0) {
    return 0.0f;
  }
  float to = v * dir < 0 ? 0 : dir * std::numeric_limits<float>::infinity();

  for (int i = 0; i < iter; i++) {
    fv = std::nextafter(fv, to);
  }

  return fv;
};

time_stat
RunLSIQueryRTSpatial(const std::shared_ptr<PlanarGraph<double>> &pgraph1,
                     const std::shared_ptr<PlanarGraph<double>> &pgraph2,
                     const BenchmarkConfig &config) {
  std::vector<double> points1_x, points1_y;
  std::vector<Edge<double>> edges1;

  std::vector<double> points2_x, points2_y;
  std::vector<Edge<double>> edges2;

  ExtractLineSegs(pgraph1, points1_x, points1_y, edges1);
  ExtractLineSegs(pgraph2, points2_x, points2_y, edges2);

  auto get_boxes = [](const std::vector<double> &points_x,
                      const std::vector<double> &points_y,
                      const std::vector<Edge<double>> &edges) {
    std::vector<rtspatial::Envelope<rtspatial::Point<float, 2>>> boxes;

    boxes.reserve(edges.size());

    for (auto &e : edges) {
      auto min_x = std::min(points_x[e.p1_idx], points_x[e.p2_idx]);
      auto min_y = std::min(points_y[e.p1_idx], points_y[e.p2_idx]);
      auto max_x = std::max(points_x[e.p1_idx], points_x[e.p2_idx]);
      auto max_y = std::max(points_y[e.p1_idx], points_y[e.p2_idx]);

      rtspatial::Envelope<rtspatial::Point<float, 2>> box(
          rtspatial::Point<float, 2>(next_float_from_double(min_x, -1, 2),
                                     next_float_from_double(min_y, -1, 2)),
          rtspatial::Point<float, 2>(next_float_from_double(max_x, 1, 2),
                                     next_float_from_double(max_y, 1, 2)));
      boxes.push_back(box);
    }
    return boxes;
  };
  thrust::device_vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>
      d_boxes = get_boxes(points1_x, points1_y, edges1),
      d_queries = get_boxes(points2_x, points2_y, edges2);

  std::cout << "Loaded\n";

  rtspatial::Stream stream;
  rtspatial::SpatialIndex<float, 2> index;
  rtspatial::Config idx_config;

  idx_config.max_geometries = d_boxes.size();
  idx_config.max_queries = d_queries.size();
  idx_config.ptx_root = std::string(RTSPATIAL_PTX_DIR);
  idx_config.intersect_cost_weight = 0.90;
  idx_config.prefer_fast_build_query = false;

  index.Init(idx_config);
  time_stat ts;
  Stopwatch sw;

  ts.num_geoms = d_boxes.size();
  ts.num_queries = d_queries.size();

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    index.Clear();
    sw.start();
    index.Insert(
        rtspatial::ArrayView<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>(
            d_boxes),
        stream.hip_stream());
    stream.Sync();
    sw.stop();
    ts.insert_ms.push_back(sw.ms());
  }

  d_boxes.resize(0);
  d_boxes.shrink_to_fit();

  rtspatial::Queue<thrust::pair<uint32_t, uint32_t>> results;
  rtspatial::SharedValue<
      rtspatial::Queue<thrust::pair<uint32_t, uint32_t>>::device_t>
      d_results;

  results.Init(std::max(
      1ul, (size_t)(ts.num_geoms * ts.num_queries * config.load_factor)));
  d_results.set(stream.hip_stream(), results.DeviceObject());

  int best_parallelism =
      index.CalculateBestParallelism(d_queries, stream.hip_stream());

  for (int i = 0; i < config.warmup + config.repeat; i++) {
    results.Clear(stream.hip_stream());
    sw.start();
    index.IntersectsWhatQuery(d_queries, d_results.data(), stream.hip_stream(),
                              best_parallelism);
    ts.num_results = results.size(stream.hip_stream());
    sw.stop();
    std::cout << sw.ms() << std::endl;
    ts.query_ms.push_back(sw.ms());
  }

  return ts;
}
