#include "hip/hip_runtime.h"
#include "query/rtspatial/common.h"
#include "query/rtspatial/update.h"
#include "rtspatial/rtspatial.h"
#include "stopwatch.h"


time_stat RunInsertionRTSpatial(const std::vector<box_t> &boxes,
                                const BenchmarkConfig &config) {
  rtspatial::Stream stream;
  rtspatial::SpatialIndex<coord_t, 2> index;
  thrust::device_vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>
      d_boxes;
  rtspatial::Config idx_config;

  idx_config.ptx_root = std::string(RTSPATIAL_PTX_DIR);
  idx_config.max_geometries = boxes.size();
  CopyBoxes(boxes, d_boxes);

  index.Init(idx_config);
  time_stat ts;
  Stopwatch sw;

  ts.num_geoms = boxes.size();

  int batch = config.batch;

  if (batch == -1) {
    size_t n_steps = 100;
    size_t avg_gemos_per_step = (boxes.size() + n_steps - 1) / n_steps;
    size_t n_inserted = 0;

    for (size_t i = 0; i < n_steps; i++) {
      auto begin = i * avg_gemos_per_step;
      auto size = std::min(begin + avg_gemos_per_step, boxes.size()) - begin;
      double total_insert_time = 0;

      n_inserted += size;

      for (int repeat = 0; repeat < config.repeat; repeat++) {
        index.Clear();

        sw.start();
        index.Insert(rtspatial::ArrayView<
                         rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>(
                         thrust::raw_pointer_cast(d_boxes.data()), n_inserted),
                     stream.hip_stream());

        stream.Sync();
        sw.stop();
        total_insert_time += sw.ms();
      }

      std::cout << "Step " << i << " Geoms " << n_inserted << " Insert Time "
                << total_insert_time / config.repeat << " ms" << std::endl;
    }
  } else {
    double total_insert_time = 0;

    size_t n_batches = (boxes.size() + batch - 1) / batch;

    for (int repeat = 0; repeat < config.repeat; repeat++) {
      index.Clear();

      sw.start();
      for (int batch_id = 0; batch_id < n_batches; batch_id++) {
        size_t batch_begin = batch_id * batch;
        size_t batch_size =
            std::min(batch_begin + batch, boxes.size()) - batch_begin;

        index.Insert(rtspatial::ArrayView<
                         rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>(
                         thrust::raw_pointer_cast(d_boxes.data()) + batch_begin,
                         batch_size),
                     stream.hip_stream());
      }
      stream.Sync();
      sw.stop();
      total_insert_time += sw.ms();
    }
    total_insert_time /= config.repeat;

    std::cout << "Batch " << batch << " Geoms " << boxes.size()
              << " Insert Time " << total_insert_time << " ms Throughput "
              << boxes.size() / (total_insert_time / 1000) << " geoms/sec"
              << std::endl;
  }
  return ts;
}

time_stat RunDeletionRTSpatial(const std::vector<box_t> &boxes,
                               const BenchmarkConfig &config) {
  rtspatial::Stream stream;
  rtspatial::SpatialIndex<coord_t, 2> index;
  thrust::device_vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>
      d_boxes;
  thrust::device_vector<size_t> deleted_ids;
  rtspatial::Config idx_config;

  idx_config.ptx_root = std::string(RTSPATIAL_PTX_DIR);
  idx_config.max_geometries = boxes.size();
  CopyBoxes(boxes, d_boxes);

  index.Init(idx_config);
  time_stat ts;
  Stopwatch sw;

  ts.num_geoms = boxes.size();

  int batch = config.batch;

  if (batch == -1) {
    size_t n_steps = 100;
    size_t avg_gemos_per_step = (boxes.size() + n_steps - 1) / n_steps;
    size_t n_inserted = 0;

    for (size_t i = 0; i < n_steps; i++) {
      auto begin = i * avg_gemos_per_step;
      auto size = std::min(begin + avg_gemos_per_step, boxes.size()) - begin;
      double total_delete_time = 0;

      n_inserted += size;

      for (int repeat = 0; repeat < config.repeat; repeat++) {
        index.Clear();

        index.Insert(rtspatial::ArrayView<
                         rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>(
                         thrust::raw_pointer_cast(d_boxes.data()), n_inserted),
                     stream.hip_stream());
        stream.Sync();

        deleted_ids.resize(n_inserted);
        thrust::transform(thrust::cuda::par.on(stream.hip_stream()),
                          thrust::make_counting_iterator<size_t>(0),
                          thrust::make_counting_iterator<size_t>(n_inserted),
                          deleted_ids.begin(), thrust::identity<size_t>());

        sw.start();
        index.Delete(rtspatial::ArrayView<size_t>(deleted_ids),
                     stream.hip_stream());
        stream.Sync();
        sw.stop();
        total_delete_time += sw.ms();
      }

      std::cout << "Step " << i << " Geoms " << n_inserted << " Delete Time "
                << total_delete_time / config.repeat << " ms" << std::endl;
    }
  } else {
    double total_delete_time = 0;
    size_t n_batches = (boxes.size() + batch - 1) / batch;

    for (int repeat = 0; repeat < config.repeat; repeat++) {
      index.Clear();

      index.Insert(rtspatial::ArrayView<
                       rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>(
                       thrust::raw_pointer_cast(d_boxes.data()), boxes.size()),
                   stream.hip_stream());

      for (int batch_id = 0; batch_id < n_batches; batch_id++) {
        size_t batch_begin = batch_id * batch;
        size_t batch_end = std::min(batch_begin + batch, boxes.size());
        size_t batch_size = batch_end - batch_begin;

        deleted_ids.resize(batch_size);
        thrust::transform(thrust::cuda::par.on(stream.hip_stream()),
                          thrust::make_counting_iterator<size_t>(batch_begin),
                          thrust::make_counting_iterator<size_t>(batch_end),
                          deleted_ids.begin(), thrust::identity<size_t>());
        sw.start();
        index.Delete(rtspatial::ArrayView<size_t>(deleted_ids),
                     stream.hip_stream());
        stream.Sync();
        sw.stop();
        total_delete_time += sw.ms();
      }
    }

    total_delete_time /= config.repeat;

    std::cout << "Batch " << batch << " Geoms " << boxes.size()
              << " Delete Time " << total_delete_time << " ms Throughput "
              << boxes.size() / (total_delete_time / 1000) << " geoms/sec"
              << std::endl;
  }
  return ts;
}

